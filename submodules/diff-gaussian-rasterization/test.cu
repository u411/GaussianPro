
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda() {
    printf("Hello from CUDA!\n");
}

int main() {
    hello_cuda<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
